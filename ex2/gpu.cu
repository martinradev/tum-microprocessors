#include <stdio.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>

__global__ void warmup(uint8_t *arr, size_t n)
{
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    arr[tid] = 1U;
}

__global__ void test(uint8_t *arr, size_t n, size_t stride, uint64_t *timer)
{
    size_t i;
    size_t j;
    uint64_t sumDeltas = 0;
    uint64_t numReads = 0;
    for (j = 0U; j < 1U; ++j)
    {
        for (i = 0U; i < n; i += stride)
        {
            uint64_t t1 = clock64();
            arr[i] += 1U;
            uint64_t t2 = clock64();
            sumDeltas += (t2-t1);
            ++numReads;
        }
    }
    sumDeltas /= numReads;
    *timer = sumDeltas;
}

int main(void)
{
    float ms;
    size_t arraySize = 1024U * 512U;
    uint8_t *gpuBuffer = NULL;
    hipError_t err = hipMalloc(&gpuBuffer, arraySize);
    if (err != hipSuccess)
    {
        printf("Failed to alloc gpu mem\n");
        return -1;
    }
    uint64_t *gpuClock;
    err = hipMalloc(&gpuClock, sizeof(*gpuClock));
    if (err != hipSuccess)
    {
        printf("Failed to alloc clock timer\n");
        return -1;
    }
    uint64_t gpuTimerOnCpu = 0U;
    hipMemcpy(gpuClock, &gpuTimerOnCpu, sizeof(*gpuClock), hipMemcpyHostToDevice);

    hipEvent_t startEvent, endEvent;
    err = hipEventCreate(&startEvent);
    if (err != hipSuccess)
    {
        printf("Failed to create start event\n");
    }
    err = hipEventCreate(&endEvent);
    if (err != hipSuccess)
    {
        printf("Failed to create end event\n");
    }

    {
        // warm-up gpu buffer
        size_t threadBlockSize = 512U;
        size_t numBlocks = (arraySize + threadBlockSize - 1U) / threadBlockSize;
        printf("Launch warm-up kernel. Num blocks: %lu, Block size: %lu\n", numBlocks, threadBlockSize);
        warmup<<<dim3(numBlocks,1,1), dim3(threadBlockSize,1,1)>>>(gpuBuffer, arraySize); 
    }

    printf("Warm up gpu cache line kernel\n");
    err = hipEventRecord(startEvent);
    test<<<dim3(1,1,1), dim3(1,1,1)>>>(gpuBuffer, arraySize, 1U, gpuClock);
    err = hipEventRecord(endEvent);
    err = hipEventSynchronize(endEvent);
    err = hipEventElapsedTime(&ms, startEvent, endEvent);
    printf("Warm-up took %f\n", ms);

    FILE *out = fopen("gpu_cache_line_size_data.txt", "w+");
    const size_t maxStrideSize = 2048U;
    for (size_t i = 1; i < maxStrideSize; ++i)
    {
        double totalms = .0f;
        for (size_t q = 0U; q < 64U; ++q)
        {
            test<<<dim3(1,1,1), dim3(1,1,1)>>>(gpuBuffer, arraySize, i, gpuClock);
            hipMemcpy(&gpuTimerOnCpu, gpuClock, sizeof(*gpuClock), hipMemcpyDeviceToHost);
            totalms += (double)gpuTimerOnCpu;
        }
        fprintf(out, "%lu %f\n", i, i*totalms/(arraySize * 64U));
        printf("Done %lu/%lu\n", i, maxStrideSize);
    }
    fclose(out);

    hipEventDestroy(endEvent);
    hipEventDestroy(startEvent);
    hipFree(gpuBuffer);

    return 0;
}
